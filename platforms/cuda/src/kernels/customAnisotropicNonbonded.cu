#include "hip/hip_runtime.h"
inline __device__ real4 delta(real4 vec1, real4 vec2, real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ) {
    real4 result = make_real4(vec1.x-vec2.x,vec1.y-vec2.y,vec1.z-vec2.z,0);
    #ifdef USE_PERIODIC
        APPLY_PERIODIC_TO_DELTA(result);
    #endif
    result.w = result.x*result.x+result.y*result.y+result.z*result.z;
    return result;
}

inline __device__ real computeDot(real4 vec1, real4 vec2) {
    real result = vec1.x*vec2.x+vec1.y*vec2.y+vec1.z*vec2.z;
    return result;
}


inline __device__ real4 computeCross(real4 vec1, real4 vec2) {
    real3 result = make_real3(vec1.y*vec2.z-vec1.z*vec2.y, vec1.z*vec2.x-vec1.x*vec2.z, vec1.x*vec2.y-vec1.y*vec2.x);
    return make_real4(result.x,result.y,result.z,result.x*result.x + result.y*result.y + result.z*result.z);
}


//Angle Calculations
inline __device__ real computeAzim(real4 vec1, real4 vec2) {
    real angle;
    real dot = computeDot(vec1,vec2);
    vec1.w = computeDot(vec1,vec1);
    vec2.w = computeDot(vec2,vec2);
    real cosine = dot/sqrt(vec1.w*vec2.w);
    if (cosine > 0.99 || cosine < -0.99) {
        real4 cp = computeCross(vec1,vec2);
        angle = asin(sqrt(cp.w/(vec1.w*vec2.w)));
        if (cosine < 0.0) angle = M_PI-angle;
    }
    else angle = acos(cosine);
    return angle;
}

inline __device__ real computePolar(real4 vec1, real4 vec2, real4 vec3) {
    real angle;
    real4 cross1 = computeCross(vec1,vec2);
    real4 cross2 = computeCross(vec2,vec3);
    real dot = computeDot(cross1,cross2);
    real4 cross3 = computeCross(cross1,cross2);

    if (dot != 0.0) dot /= sqrt(cross1.w*cross2.w);
    if (dot > 1.0) dot = 1.0;
    else if (dot < -1.0) dot = -1.0;

    if (dot > 0.99 || dot < -0.99) {
        angle = asin(sqrt(cross3.w/(cross1.w*cross2.w)));
        if (dot < 0.0) angle = M_PI - angle;
    }
    else angle = acos(dot);
    return angle;
}

extern "C" __global__ void accessAxisParameter(const real4* __restrict__ posq, const int4* __restrict__ axes, real4* __restrict__ kvecs) {
    real4 vectorX,vectorY,vectorZ,vectemp;
    int axisType,pX,pY,pZ;
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom< NUM_ATOMS; atom += gridDim.x*blockDim.x) {
        __syncthreads();
        axisType = axes[atom].w;
        pX = axes[atom].x;
        pY = axes[atom].y;
        pZ = axes[atom].z;
        if (axisType == 5) {
            vectorZ = make_real4(0.0,0.0,1.0,1.0);
            vectorX = make_real4(1.0,0.0,0.0,1.0);
        }
        else {
            vectorZ = posq[pZ]-posq[atom];
            if (axisType == 4) {
                if (fabs(vectorZ.x) < 0.866f) vectorX = make_real4(1.0,0.0,0.0,1.0);
                else vectorX = make_real4(0.0,1.0,0.0,1.0);
            }
            else {
                vectorY = posq[pY]-posq[atom];
                if (axisType == 1) {
                    vectemp = computeCross(vectorZ,vectorY);
                    vectorZ += vectorY;
                    vectorZ *= 0.5;
                    vectorX = vectemp;
                }
                else {
                    vectorX = posq[pX]-posq[atom];
                    if (axisType == 2) {
                        vectorX += vectorY;
                        vectorX *= 0.5;
                    }
                    else if (axisType == 3) {
                        vectorZ += vectorX+vectorY;
                        vectorZ *= 1/3.0;
                        if (fabs(vectorZ.x) < 0.866f) vectorX = make_real4(1.0,0.0,0.0,1.0);
                        else vectorX = make_real4(0.0,1.0,0.0,1.0);
                    }
                }
            }
        }
        vectorX.w = computeDot(vectorX,vectorX);
        vectorZ.w = computeDot(vectorZ,vectorZ);
        vectorY = computeCross(vectorZ,vectorX);
        kvecs[atom*3] = vectorX;
        kvecs[atom*3+1] = vectorY;
        kvecs[atom*3+2] = vectorZ;
    }
}

extern "C" __global__ void computeCAN(unsigned long long* __restrict__ force, mixed* __restrict__ energyBuffer, const real4* __restrict__ posq, 
    const int4* __restrict__ axes, real4* __restrict__ kvecs,
    real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ 
    #ifdef USE_EXCLUSIONS
    , int* __restrict__ exclusions, int* __restrict exclusionStartIdx
    #endif
    PARAMETER_ARGUMENTS) {
    real dEdR=0.0; real dEdTheta1=0.0; real dEdTheta2=0.0; real dEdPhi1=0.0; real dEdPhi2=0.0;
    real4 f1=make_real4(0.0); real4 f2 = make_real4(0.0); real4 f3 = make_real4(0.0); real4 f4 = make_real4(0.0); real4 f0 = make_real4(0.0);
    int4 axs2, axs1;
    real4 kvec1x,kvec1y,kvec1z,kvec2x,kvec2y,kvec2z;
    mixed energy=0.0;


    int ii =  blockIdx.x*blockDim.x+threadIdx.x;
    if (ii < NUM_ATOMS) {
    axs1 = axes[ii]; 
    kvec1x = kvecs[ii*3];
    kvec1y = kvecs[ii*3+1];
    kvec1z = kvecs[ii*3+2];

    for (int jj = ii+1; jj < NUM_ATOMS; jj += 1) {
        __syncthreads();

        bool isExcluded=0;
        #ifdef USE_EXCLUSIONS
        int check = 0;
        int first = exclusionStartIdx[ii];
        int last = exclusionStartIdx[ii+1];
        for (int ex = last-1; ex >= first; ex--) {
            if (exclusions[ex] == jj) check+=1;
            if (check!=0) break;
        }
        if (check!=0) isExcluded = 1;
        #endif
        if (!isExcluded) {
            axs2 = axes[jj];
            kvec2x = kvecs[jj*3];
            kvec2y = kvecs[jj*3+1];
            kvec2z = kvecs[jj*3+2];
            //ixn calc here
            real4 rij = delta(posq[jj],posq[ii],periodicBoxSize,invPeriodicBoxSize,periodicBoxVecX,periodicBoxVecY,periodicBoxVecZ);
            #ifdef USE_CUTOFF
            if (rij.w < CUTOFF_SQUARED) {
            #endif
            real r = sqrt(rij.w);
            COMPUTE_FORCE
            f0 = dEdR*rij/r;
            //write results here
            //distance dependent forces
            atomicAdd(&force[ii],static_cast<unsigned long long>((long long) (f0.x*0x100000000)));
            atomicAdd(&force[ii+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f0.y*0x100000000)));
            atomicAdd(&force[ii+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f0.z*0x100000000)));
            atomicAdd(&force[jj],static_cast<unsigned long long>((long long) (-f0.x*0x100000000)));
            atomicAdd(&force[jj+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f0.y*0x100000000)));
            atomicAdd(&force[jj+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f0.z*0x100000000)));
            //angle expressions here
            //thetas
            if (dEdTheta1*dEdTheta1 > 0.0) {
                real4 thcross = computeCross(-kvec1z,-rij);
                real thcrossL = sqrt(thcross.w);
                if (thcrossL < 1.0e-06) thcrossL = 1.0e-06;
                real termA = dEdTheta1/(kvec1z.w*thcrossL);
                real termC = -dEdTheta1/(rij.w*thcrossL);
                f1 = termA*computeCross(-kvec1z,thcross);
                f3 = termC*computeCross(-rij,thcross);
                f2 = -f1-f3;
                //root atoms
                atomicAdd(&force[ii],static_cast<unsigned long long>((long long) (f2.x*0x100000000)));
                atomicAdd(&force[ii+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f2.y*0x100000000)));
                atomicAdd(&force[ii+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f2.z*0x100000000)));
                atomicAdd(&force[jj],static_cast<unsigned long long>((long long) (f3.x*0x100000000)));
                atomicAdd(&force[jj+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.y*0x100000000)));
                atomicAdd(&force[jj+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.z*0x100000000)));
                __threadfence_block();
                //axis atoms
                if(axs1.w == 4 || axs1.w == 0 || axs1.w == 2) {
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs1.w == 1) {
                    f1 = 0.5*f1;
                    atomicAdd(&force[axs1.y],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs1.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs1.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs1.w == 3) {
                    f1 = f1/3.0;
                    atomicAdd(&force[axs1.x],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs1.x+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs1.x+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                    atomicAdd(&force[axs1.y],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs1.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs1.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                __threadfence_block();
                }
            }
            if (dEdTheta2*dEdTheta2 > 0.0) {
                real4 thcross = computeCross(-kvec2z,rij);
                real thcrossL = sqrt(thcross.w);
                if (thcrossL < 1.0e-06) thcrossL = 1.0e-06;
                real termA = dEdTheta2/(kvec2z.w*thcrossL);
                real termC = -dEdTheta2/(rij.w*thcrossL);
                f1 = termA*computeCross(-kvec2z,thcross);
                f3 = termC*computeCross(rij,thcross);
                f2 = -f1-f3;
                //root atoms
                atomicAdd(&force[jj],static_cast<unsigned long long>((long long) (f2.x*0x100000000)));
                atomicAdd(&force[jj+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f2.y*0x100000000)));
                atomicAdd(&force[jj+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f2.z*0x100000000)));
                atomicAdd(&force[ii],static_cast<unsigned long long>((long long) (f3.x*0x100000000)));
                atomicAdd(&force[ii+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.y*0x100000000)));
                atomicAdd(&force[ii+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.z*0x100000000)));
                __threadfence_block();
                //axis atoms
                if(axs2.w == 4 || axs2.w == 0 || axs2.w == 2) {
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs2.w == 1) {
                    f1 = 0.5*f1;
                    atomicAdd(&force[axs2.y],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs2.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs2.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs2.w == 3) {
                    f1 = f1/3.0;
                    atomicAdd(&force[axs2.x],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs2.x+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs2.x+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                    atomicAdd(&force[axs2.y],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs2.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs2.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                __threadfence_block();
                }
            }
            if (dEdPhi1*dEdPhi1 > 0.0) {
                real4 cross2 = computeCross(kvec1z,kvec1z-rij);
                if (cross2.w < 1.0e-06) cross2.w = 1.0e-06;
                real normBC = sqrt(kvec1z.w);
                f1 = dEdPhi1*normBC*kvec1y/kvec1y.w;
                f4 = dEdPhi1*normBC*cross2/cross2.w;
                real dot1 = computeDot(kvec1x,kvec1z)/kvec1z.w;
                real dot2 = computeDot(kvec1z-rij,kvec1z)/kvec1z.w;
                real4 ss = dot1*f1-dot2*f4;
                f2 = f1-ss;
                f3 = f4+ss; 
                atomicAdd(&force[ii],static_cast<unsigned long long>((long long) (-f2.x*0x100000000)));
                atomicAdd(&force[ii+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f2.y*0x100000000)));
                atomicAdd(&force[ii+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f2.z*0x100000000)));
                atomicAdd(&force[jj],static_cast<unsigned long long>((long long) (f4.x*0x100000000)));
                atomicAdd(&force[jj+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f4.y*0x100000000)));
                atomicAdd(&force[jj+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f4.z*0x100000000)));
                __threadfence_block();
                if (axs1.w == 0) {
                    atomicAdd(&force[axs1.x],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs1.x+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs1.x+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (-f3.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs1.w == 1) {
                    atomicAdd(&force[axs1.y],static_cast<unsigned long long>((long long) (-0.5*f3.x*0x100000000)));
                    atomicAdd(&force[axs1.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-0.5*f3.y*0x100000000)));
                    atomicAdd(&force[axs1.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-0.5*f3.z*0x100000000)));
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (-0.5*f3.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-0.5*f3.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-0.5*f3.z*0x100000000)));
                    real4 f5 = computeCross(posq[axs1.z]-posq[ii],f1);
                    real4 f6 = computeCross(f1,posq[axs1.y]-posq[ii])+0.5*f1;
                    atomicAdd(&force[axs1.y],static_cast<unsigned long long>((long long) (f5.x*0x100000000)));
                    atomicAdd(&force[axs1.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f5.y*0x100000000)));
                    atomicAdd(&force[axs1.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f5.z*0x100000000)));
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (f6.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f6.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f6.z*0x100000000)));
                    atomicAdd(&force[ii],static_cast<unsigned long long>((long long) ((f1.x-f5.x-f6.x)*0x100000000)));
                    atomicAdd(&force[ii+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) ((f1.y-f5.y-f6.y)*0x100000000)));
                    atomicAdd(&force[ii+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) ((f1.z-f5.z-f6.z)*0x100000000)));
                __threadfence_block();
                }
                else if (axs1.w == 2) {
                    atomicAdd(&force[axs1.x],static_cast<unsigned long long>((long long) (0.5*f1.x*0x100000000)));
                    atomicAdd(&force[axs1.x+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (0.5*f1.y*0x100000000)));
                    atomicAdd(&force[axs1.x+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (0.5*f1.z*0x100000000)));
                    atomicAdd(&force[axs1.y],static_cast<unsigned long long>((long long) (0.5*f1.x*0x100000000)));
                    atomicAdd(&force[axs1.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (0.5*f1.y*0x100000000)));
                    atomicAdd(&force[axs1.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (0.5*f1.z*0x100000000)));
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (-f3.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs1.w == 3) {
                    f3 = -f3/3.0; 
                    atomicAdd(&force[axs1.x],static_cast<unsigned long long>((long long) (f3.x*0x100000000)));
                    atomicAdd(&force[axs1.x+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.y*0x100000000)));
                    atomicAdd(&force[axs1.x+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.z*0x100000000)));
                    atomicAdd(&force[axs1.y],static_cast<unsigned long long>((long long) (f3.x*0x100000000)));
                    atomicAdd(&force[axs1.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.y*0x100000000)));
                    atomicAdd(&force[axs1.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.z*0x100000000)));
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (f3.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs1.w == 4) {
                    atomicAdd(&force[axs1.z],static_cast<unsigned long long>((long long) (-f3.x*0x100000000)));
                    atomicAdd(&force[axs1.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.y*0x100000000)));
                    atomicAdd(&force[axs1.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.z*0x100000000)));
                __threadfence_block();
                }
            }
            if (dEdPhi2*dEdPhi2 > 0.0) {
                real4 cross2 = computeCross(kvec2z,kvec2z+rij);
                if (cross2.w < 1.0e-06) cross2.w = 1.0e-06;
                real normBC = sqrt(kvec2z.w);
                f1 = dEdPhi2*normBC*kvec2y/kvec2y.w;
                f4 = dEdPhi2*normBC*cross2/cross2.w;
                real dot1 = computeDot(kvec2x,kvec2z)/kvec2z.w;
                real dot2 = computeDot(kvec2z+rij,kvec2z)/kvec2z.w;
                real4 ss = dot1*f1-dot2*f4;
                f2 = f1-ss;
                f3 = f4+ss;
                atomicAdd(&force[jj],static_cast<unsigned long long>((long long) (-f2.x*0x100000000)));
                atomicAdd(&force[jj+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f2.y*0x100000000)));
                atomicAdd(&force[jj+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f2.z*0x100000000)));
                atomicAdd(&force[ii],static_cast<unsigned long long>((long long) (f4.x*0x100000000)));
                atomicAdd(&force[ii+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f4.y*0x100000000)));
                atomicAdd(&force[ii+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f4.z*0x100000000)));
                __threadfence_block();
                if (axs2.w == 0) {
                    atomicAdd(&force[axs2.x],static_cast<unsigned long long>((long long) (f1.x*0x100000000)));
                    atomicAdd(&force[axs2.x+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.y*0x100000000)));
                    atomicAdd(&force[axs2.x+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f1.z*0x100000000)));
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (-f3.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs2.w == 1) {
                    atomicAdd(&force[axs2.y],static_cast<unsigned long long>((long long) (-0.5*f3.x*0x100000000)));
                    atomicAdd(&force[axs2.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-0.5*f3.y*0x100000000)));
                    atomicAdd(&force[axs2.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-0.5*f3.z*0x100000000)));
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (-0.5*f3.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-0.5*f3.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-0.5*f3.z*0x100000000)));
                    real4 f5 = computeCross(posq[axs2.z]-posq[jj],f1);
                    real4 f6 = computeCross(f1,posq[axs2.y]-posq[jj])+0.5*f1;
                    atomicAdd(&force[axs2.y],static_cast<unsigned long long>((long long) (f5.x*0x100000000)));
                    atomicAdd(&force[axs2.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f5.y*0x100000000)));
                    atomicAdd(&force[axs2.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f5.z*0x100000000)));
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (f6.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f6.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f6.z*0x100000000)));
                    atomicAdd(&force[jj],static_cast<unsigned long long>((long long) ((f1.x-f5.x-f6.x)*0x100000000)));
                    atomicAdd(&force[jj+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) ((f1.y-f5.y-f6.y)*0x100000000)));
                    atomicAdd(&force[jj+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) ((f1.z-f5.z-f6.z)*0x100000000)));
                __threadfence_block();
                }
                else if (axs2.w == 2) {
                    atomicAdd(&force[axs2.x],static_cast<unsigned long long>((long long) (0.5*f1.x*0x100000000)));
                    atomicAdd(&force[axs2.x+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (0.5*f1.y*0x100000000)));
                    atomicAdd(&force[axs2.x+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (0.5*f1.z*0x100000000)));
                    atomicAdd(&force[axs2.y],static_cast<unsigned long long>((long long) (0.5*f1.x*0x100000000)));
                    atomicAdd(&force[axs2.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (0.5*f1.y*0x100000000)));
                    atomicAdd(&force[axs2.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (0.5*f1.z*0x100000000)));
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (-f3.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs2.w == 3) {
                    f3 = -f3/3.0;
                    atomicAdd(&force[axs2.x],static_cast<unsigned long long>((long long) (f3.x*0x100000000)));
                    atomicAdd(&force[axs2.x+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.y*0x100000000)));
                    atomicAdd(&force[axs2.x+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.z*0x100000000)));
                    atomicAdd(&force[axs2.y],static_cast<unsigned long long>((long long) (f3.x*0x100000000)));
                    atomicAdd(&force[axs2.y+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.y*0x100000000)));
                    atomicAdd(&force[axs2.y+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.z*0x100000000)));
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (f3.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (f3.z*0x100000000)));
                __threadfence_block();
                }
                else if (axs2.w == 4) {
                    atomicAdd(&force[axs2.z],static_cast<unsigned long long>((long long) (-f3.x*0x100000000)));
                    atomicAdd(&force[axs2.z+PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.y*0x100000000)));
                    atomicAdd(&force[axs2.z+2*PADDED_NUM_ATOMS],static_cast<unsigned long long>((long long) (-f3.z*0x100000000)));
                __threadfence_block();
                }
            }
            #ifdef USE_CUTOFF
            }
            #endif
        }
        }
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
}
