#include "hip/hip_runtime.h"
//CustomAnisotropicNonbonded

//Define useful functions.. first 4 in CustomHbond
//Vector simplification
inline __device__ real3 trim(real4 v) {
	return make_real3(v.x,v.y,v.z);
}
inline __device__ real3 trim(real3 v) {
	return v;
}
//Periodic and nonperiodic difference
inline __device__ real4 delta(real4 vec1, real4 vec2, real4 periodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ) {
	real4 result = make_real4(vec1.x-vec2.x,vec1.y-vec2.y,vec1.z-vec2.z,0.0f);
	#ifdef USE_PERIODIC
		APPLY_PERIODIC_TO_DELTA(result)
	#endif
	result.w = result.x*result.x+result.y*result.y+result.z*result.z;
	return result
}
inline __device__ real4 computeCross(real4 vec1, real4 vec2) {
	real3 result = cross(vec1,vec2);
	return make_real4(result.x,result.y,result.z,result.x*result.x + result.y*result.y + result.z*result.z)
}

//Angle Calculations
inline __device__ real computePhi(real4 vec1, real4 vec2) {
	real angle;
	real dot = vec1.x*vec2.x + vec1.y*vec2.y + vec1.z*vec2.z;
	double cosine = dot/RSQRT(vec1.w*vec2.w);
	if (cosine >= 1) angle = 0;
	else if (cosine <= -1) angle = M_PI;
	else angle = acos(cosine);
	return angle
}
inline __device__ real computeTheta(real4 vec1, real4 vec2, real4 vec3) {
	real angle;
	real4 cross1 = computeCross(vec1,vec2);
	real4 cross2 = computeCross(vec2,vec3);
	real dot = cross1.x*cross2.x+cross1.y*cross2.y+cross1.z*cross2.z;
	real4 cross = computeCross(cross1,cross2);

	if (dot != 0.0) dot /= RSQRT(cross1.w*cross2.w);
	if (dot > 1.0) dot = 1.0;
	else if (dot < -1.0) dot = -1.0;

	if (dot > 0.99 || dot < -0.99) {
		angle = ASIN(RSQRT(cross.w/(cross1.w*cross2.w)));
		if (dot < 0.0) angle = M_PI - angle;
	}
	else angle = ACOS(dot);
	return angle;
}

inline __device__ void accessAxisParameter(real4* pos, int4 kparticles, real3* kvecs) {
	real3 vectorX,vectorY,vectorZ,vectemp;
	int axisType,pX,pY,pZ;
	for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom< NUM_ATOMS; atom += gridDim.x*blockDim.x) {
		axisType = kparticles.w;
		pX = kparticles.x;
		pY = kparticles.y;
		pZ = kparticles.z;
		if (axisType == CustomAnisotropicNonbondedForce::NoAxisType) {
			vectorZ = make_real3(0,0,1);
			vectorX = make_real3(1,0,0);
		}
		else {
			vectorZ = pos[pZ] - pos[atom];
			if (axisType == CustomAnisotropicNonbondedForce::ZOnly) {
				if (ABS(vectorZ.x) < 0.866) vectorX = make_real3(1.0,0.0,0.0);
				else vectorX = make_real3(0.0,1.0,0.0);
			}
			else {
				vectorX = pos[pX] - pos[atom];
				if (axisType == CustomAnisotropicNonbondedForce::Bisector) {
					vectemp = cross(vectorZ,vectorX);
					vectorZ += vectorX;
					vector *= 0.5;
					vectorX = vectemp;
				}
				else {
					vectorY = pos[pY] - pos[atom];
					if (axisType == CustomAnisotropicNonbondedForce::ZBisect) {
						vectorX += vectorY;
						vectorX *= 0.5;
					}
					else if (axisType == CustomAnisotropicNonbondedForce::ThreeFold) {
						vectorZ += vectorX + vectorY;
						vectorZ *= 1/3.0;
						if (ABS(vectorZ.x) < 0.866) vectorX = make_real3(1.0,0.0,0.0);
						else vectorX = make_real3(0.0,1.0,0.0);
					}
				}
			}
		}
		vectorY = cross(vectorZ,vectorX);
		kvecs[atom] = vectorX;
		kvecs[atom*3+1] = vectorY;
		kvecs[atom*3+2] = vectorZ;
	}
}


/*
#ifdef USE_CUTOFF
if (!isExcluded && r2 < CUTOFF_SQUARED) {
#else
if (!isExcluded) {
#endif

	real tempForce = 0;
	reala switchValue = 1, switchDeriv = 0;
#if USE_SWITCH
	if (r > SWITCH_CUTOFF) {
		real x = r - SWITCH_CUTOFF;
		switchValue = 1+x*x*x*(SWITCH_C3+x*(SWITCH_C4+x*SWITCH_C5));
		switchDeriv = x*x*(3*SWITCH_C3+x*(4*SWITCH_C4+x*5*SWITCH_C5));
	}
#endif
	COMPTE_FORCE
#if USE_SWITCH
	tempForce = tempForce*switchValue - tempEnergy*switchDeriv;
	tempEnergy *= switchValue;
#endif
	dEdR += tempForce*invR;
}
*/
